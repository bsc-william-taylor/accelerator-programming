
#include "gpu_kernels.h"
#include "gpu_memory.h"

#include <iostream>
#include <locale>

void cudaCheckError()
{
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    std::cin.get();
}

int main(void)
{  
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);

    //std::cout << props.name << std::endl;
    //std::cout << props.clockRate << std::endl;
    //std::cin.get();
    
    int numbers1[2024];
    int numbers2[2024];
    int numbers3[2024];

    int* deviceNumbers1, *deviceNumbers2, *deviceNumbers3;

    memset(numbers1, 10, sizeof(numbers1));
    memset(numbers2, 50, sizeof(numbers2));
    memset(numbers3, 0, sizeof(numbers3));

    hipMalloc((void**)&deviceNumbers1, sizeof(numbers1));
    hipMalloc((void**)&deviceNumbers2, sizeof(numbers1));
    hipMalloc((void**)&deviceNumbers3, sizeof(numbers1));

    hipMemcpy(deviceNumbers1, numbers1, sizeof(numbers1), hipMemcpyHostToDevice);
    hipMemcpy(deviceNumbers2, numbers2, sizeof(numbers2), hipMemcpyHostToDevice);
    hipMemcpy(deviceNumbers3, numbers3, sizeof(numbers3), hipMemcpyHostToDevice);
    
    //cudaCheckError();
    
    vector_add<<<2024, 1>>>(deviceNumbers1, deviceNumbers2, deviceNumbers3);

    hipMemcpy(numbers3, deviceNumbers3, sizeof(numbers3), hipMemcpyDeviceToHost);

    std::cout << numbers3[0] << std::endl;
    std::cin.get();

    hipFree(deviceNumbers1);
    hipFree(deviceNumbers2);
    hipFree(deviceNumbers3);
    return 0;
}