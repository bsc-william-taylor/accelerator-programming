#include "hip/hip_runtime.h"

#include "cuda-launch.h"
#include <stdexcept>

cuda::launchInfo cuda::optimumLaunch(void* kernel, int dataLength)
{
    auto minGridSize = 0, blockSize = 0;
    auto hipError_t = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, dataLength);

    if(hipError_t != 0)
    {
        throw std::runtime_error("hipOccupancyMaxPotentialBlockSize failed");
    }
    
    const auto gridSize = (dataLength + blockSize - 1) / blockSize;

    return { gridSize, blockSize, static_cast<int>(sqrt(dataLength)) };
}
