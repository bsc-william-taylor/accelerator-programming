#include "hip/hip_runtime.h"
#include "cuda-utilities.h"
#include <cstdio>
#include <hip/hip_runtime.h>

struct rgb
{
    unsigned char r, g, b;
};

using uchar = unsigned char;
using uint = unsigned int;

__constant__ const uchar MaxIterations = std::numeric_limits<uchar>::max();
__constant__ const uint MappingsLength = 16;
__constant__ const double CenterX = -0.6;
__constant__ const double CenterY = 0.0;
__constant__ const rgb Mappings[MappingsLength]
{
    { 66,  30,   15 },
    { 25,   7,   26 },
    { 9,    1,   47 },
    { 4,    4,   73 },
    { 0,    7,   100 },
    { 12,   44,  138 },
    { 24,   82,  177 },
    { 57,   125, 209 },
    { 134,  181, 229 },
    { 211,  236, 248 },
    { 241,  233, 191 },
    { 248,  201, 95 },
    { 255,  170, 0 },
    { 204,  128, 0 },
    { 153,  87,  0 },
    { 106,  52,  3 }
};

__global__ void mandelbrot(cuda::launchInfo info, rgb* image, double scale)
{
    auto workload = cuda::allocateWork(info, blockIdx.x, threadIdx.x);
    auto px = image + workload.offset;

    for(auto i = workload.offset; i < workload.size; ++i, ++px)
    {
        auto x = (i % info.width - info.width / 2) * scale + CenterX;
        auto y = (i / info.width - info.height / 2) * scale + CenterY;
        auto zx = hypot(x - 0.25, y), zy = 0.0, zx2 = 0.0, zy2 = 0.0;

        uchar iterations = 255 * (x < zx - 2 * zx * zx + 0.2);
        
        if (x < zx - 2 * zx * zx + .25)
        {
            iterations = MaxIterations;
        }
        else if((x + 1)*(x + 1) + y * y < 1 / 16)
        {
            iterations = MaxIterations;
        }
        else
        {
            do
            {
                zy = 2 * zx * zy + y;
                zx = zx2 - zy2 + x;
                zx2 = zx * zx;
                zy2 = zy * zy;
            } while (iterations++ < MaxIterations && zx2 + zy2 < 4);
        }

        if (iterations == MaxIterations || iterations == 0)
        {
            *px = { 0 };
        }
        else
        {
            *px = Mappings[iterations % MappingsLength];
        }
    }
}

void writeOutput(const std::string& filename, rgb* image, int width, int height)
{
    auto file = fopen(filename.c_str(), "w");
        
    if(file != nullptr)
    {
        fprintf(file, "P6\n%d %d\n255\n", width, height);

        for (auto i = height - 1; i >= 0; --i) 
        {
            fwrite(image + i * width, 1, width * sizeof(rgb), file);
        }

        fclose(file);
    }
}

#include "../benchmark.h"

int main(int argc, char *argv[])
{
    benchmark<measure_in::ms, 10>([&]()
    {
        const auto height = 4096, width = 4096, threads = 64, blocks = 64;
        const auto scale = 1.0 / (width / 4);

        std::vector<rgb> image(height * width);

        cuda::launchInfo launchInfo{ blocks, threads, width, height };
        cuda::memory<rgb*> imagePointer{ image.data(), image.size() * sizeof(rgb) };
        cuda::start(mandelbrot, launchInfo, imagePointer, scale);   
        cuda::move(imagePointer, image.data());

        writeOutput("output.ppm", image.data(), width, height);
    });
    
    return 0;
}