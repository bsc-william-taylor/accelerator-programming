#include "hip/hip_runtime.h"
#include "cuda-utilities.h"
#include <cstdio>
#include "../benchmark.h"

struct rgb
{
    unsigned char r, g, b;
};

using uchar = unsigned char;
using uint = unsigned int;

__constant__ const uchar MaxIterations = std::numeric_limits<uchar>::max();
__constant__ const uint MappingsLength = 16;
__constant__ const double CenterX = -0.6;
__constant__ const double CenterY = 0.0;
__constant__ const rgb Mappings[MappingsLength]
{
    { 66,  30,   15 },
    { 25,   7,   26 },
    { 9,    1,   47 },
    { 4,    4,   73 },
    { 0,    7,   100 },
    { 12,   44,  138 },
    { 24,   82,  177 },
    { 57,   125, 209 },
    { 134,  181, 229 },
    { 211,  236, 248 },
    { 241,  233, 191 },
    { 248,  201, 95 },
    { 255,  170, 0 },
    { 204,  128, 0 },
    { 153,  87,  0 },
    { 106,  52,  3 }
};

__global__ void mandelbrot(cuda::launchInfo info, rgb* image, double scale)
{
    const auto i = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
    const auto x = (i % info.size - info.size / 2) * scale + CenterX;
    const auto y = (i / info.size - info.size / 2) * scale + CenterY;

    uchar iter = 0;

    auto zy = 0.0, zx2 = 0.0, zy2 = 0.0;
    auto zx = hypot(x - .25, y);

    if (x < zx - 2 * zx * zx + .25) 
    {
        iter = MaxIterations;
    }
    else if ((x + 1)*(x + 1) + y * y < 1 / 16)
    {
        iter = MaxIterations;
    }
    else
    {
        do 
        {
            zy = 2 * zx * zy + y;
            zx = zx2 - zy2 + x;
            zx2 = zx * zx;
            zy2 = zy * zy;
        } while (iter++ < 255 && zx2 + zy2 < 4);
    }


    if (iter == MaxIterations || iter == 0)
    {
        image[i] = { 0 };
    }
    else
    {
        image[i] = Mappings[iter % MappingsLength];
    }
}

void writeOutput(const std::string& filename, rgb* image, int width, int height)
{
    auto file = fopen(filename.c_str(), "w");
        
    if(file != nullptr)
    {
        fprintf(file, "P6\n%d %d\n255\n", width, height);

        for (auto i = height - 1; i >= 0; --i) 
        {
            fwrite(image + i * width, 1, width * sizeof(rgb), file);
        }

        fclose(file);
    }
}

int main(int argc, char *argv[])
{
    const auto height = 4096, width = 4096;
    const auto scale = 1.0 / (width / 4.0);

    std::vector<rgb> image(height * width);

    cuda::launchInfo launchInfo = optimumLaunch(mandelbrot, image.size());
    cuda::memory<rgb*> imagePointer{ image.data(), image.size() * sizeof(rgb) };
    cuda::start(mandelbrot, launchInfo, imagePointer, scale);
    cuda::move(imagePointer, image.data());

    writeOutput("output.ppm", image.data(), width, height);   
    return 0;
}